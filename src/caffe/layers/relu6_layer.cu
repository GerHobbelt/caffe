#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include <>

#include "caffe/layers/relu6_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ReLU6Forward(const int n, const Dtype* in, Dtype* out, float negative_slope) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = !signbit(in[index]) ? (in[index] > Dtype(6) ? Dtype(6) : in[index]) : Dtype(in[index] * negative_slope);
  }
}

template <typename Dtype>
__global__ void ReLU6Forward0(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = !signbit(in[index]) ? (in[index] > Dtype(6) ? Dtype(6) : in[index]) : Dtype(0);
  }
}

template <typename Ftype, typename Btype>
void ReLU6Layer<Ftype, Btype>::Forward_gpu(const vector<Blob*>& bottom,
    const vector<Blob*>& top) {
  const Ftype* bottom_data = bottom[0]->gpu_data<Ftype>();
  Ftype* top_data = top[0]->mutable_gpu_data<Ftype>();

  const int count = bottom[0]->count();
  float negative_slope = this->layer_param_.relu_param().negative_slope();
  if (negative_slope != 0.F) {
    // NOLINT_NEXT_LINE(whitespace/operators)
    ReLU6Forward <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS, 0, Caffe::thread_stream()>>>(
        count, bottom_data, top_data, negative_slope);
  } else {
    // NOLINT_NEXT_LINE(whitespace/operators)
    ReLU6Forward0 <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS, 0, Caffe::thread_stream()>>>(
        count, bottom_data, top_data);
  }
  CUDA_POST_KERNEL_CHECK;
  CUDA_CHECK(hipStreamSynchronize(Caffe::thread_stream()));
}

template <typename Ftype, typename Btype>
void ReLU6Layer<Ftype, Btype>::Backward_gpu(const vector<Blob*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob*>& bottom) {
    NOT_IMPLEMENTED;
}

INSTANTIATE_LAYER_GPU_FUNCS_FB(ReLU6Layer);

}  // namespace caffe
