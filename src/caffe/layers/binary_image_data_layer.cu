
#include <hip/hip_runtime.h>
#ifdef USE_HDF5
/*
TODO:
- only load parts of the file, in accordance with a prototxt param "max_mem"
*/

#include <stdint.h>
#include <vector>

#include "caffe/layers/binary_image_data_layer.hpp"

namespace caffe {

template <typename Dtype>
void BinaryImageDataLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {

    const int batch_size = this->layer_param_.hdf5_data_param().batch_size();
    for (int i = 0; i < batch_size; ++i)
    {
        if (current_row_ == length_)
        {
            std::random_shuffle(data_permutation_.begin(), data_permutation_.end());
            current_row_ = 0;
        }

        int data_row = data_permutation_[current_row_];
        int blob_index = std::floor(1.0 * data_row / count_per_blob_);
        int index_in_blob = data_row - blob_index * count_per_blob_;

        int size = width_in_ * height_in_;
        caffe_copy(size, &data_blobs_[0][blob_index]->cpu_data()[index_in_blob * size], 
          &top[0]->mutable_gpu_data()[i * size]);
        size = width_out_ * height_out_;
        caffe_copy(size, &data_blobs_[1][blob_index]->cpu_data()[index_in_blob * size], 
          &top[1]->mutable_gpu_data()[i * size]);

        ++current_row_;
    }
  }

INSTANTIATE_LAYER_GPU_FUNCS(BinaryImageDataLayer);

}  // namespace caffe
#endif  // USE_HDF5
