
#include <hip/hip_runtime.h>
#ifdef USE_HDF5
/*
TODO:
- only load parts of the file, in accordance with a prototxt param "max_mem"
*/

#include <stdint.h>
#include <vector>

#include "caffe/layers/binary_image_data_layer.hpp"

namespace caffe {

template <typename Dtype>
void BinaryImageDataLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {

    const int batch_size = this->layer_param_.hdf5_data_param().batch_size();
    for (int i = 0; i < batch_size; ++i)
    {
        if (current_row_ == data_.size())
        {
            std::random_shuffle(data_permutation_.begin(), data_permutation_.end());
            current_row_ = 0;
        }

        int size = width_in_ * height_in_;
        caffe_copy<Dtype>(size, data_[data_permutation_[current_row_]]->GetInput(), 
          &top[0]->mutable_gpu_data()[i * size]);
        size = width_out_ * height_out_;
        caffe_copy<Dtype>(size, data_[data_permutation_[current_row_]]->GetLabel(), 
          &top[1]->mutable_gpu_data()[i * size]);

        ++current_row_;
    }
  }

INSTANTIATE_LAYER_GPU_FUNCS(BinaryImageDataLayer);

}  // namespace caffe
#endif  // USE_HDF5
